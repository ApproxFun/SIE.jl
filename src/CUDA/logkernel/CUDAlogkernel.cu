#include "hip/hip_runtime.h"
#include "../cuda_complex/cuda_complex.hpp"

extern "C"
{

__global__ void CUDAlogkernel(const double a, const double b, const int nu, const double *u, double *x, double *y, double *ret)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int n = sizeof(x)/sizeof(x[0]);

    const double pi = M_PI;
    const double lengthd = abs(b-a);
    const double C = 0.5*lengthd;

    complex<double> *z,*yv,*yk,*ykp1;

    z = new complex<double>[n];
    yv = new complex<double>[n];
    yk = new complex<double>[n];
    ykp1 = new complex<double>[n];


    z[i] = complex<double>::complex(x[i],y[i]);
    z[i] = (a + b - 2.0*z[i])/(a - b);  // tocanonical(u,z)

    yv[i] = z[i] - sqrt(z[i]-1.0)*sqrt(z[i]+1.0);  // updownjoukowskyinverse(true,z)
    yk[i] = yv[i];
    ykp1[i] = yk[i]*yk[i];


    if ( nu >= 0 ) {
        ret[i] = -u[0]*log(abs(2.0*yk[i]/C));  // -logabs(2y/C)
        if ( nu >= 1 ) {
            ret[i] += -u[1]*real(yk[i]);  // -real(yk)
            if ( nu >= 2 ) {
                ret[i] += u[2]*(log(abs(2.0*yk[i]/C))-0.5*real(ykp1[i])); // -ret[1]-.5real(ykp1)
                if ( nu >= 3) {
                    for (int nun = 3; nun<nu; nun++) {
                        ykp1[i] *= yv[i];
                        ret[i] += u[nun]*( real(yk[i])/(nun-2.0)-real(ykp1[i])/(nun-0.0) ); // real(yk)/(n-3)-real(ykp1)/(n-1)
                        yk[i] *= yv[i];
                    }
                }
            }
        }
    }
    ret[i] *= pi*C;
}

} // extern "C"